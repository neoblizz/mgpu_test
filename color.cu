#pragma GCC diagnostic ignored "-Wunused-result"
#pragma GCC diagnostic ignored "-Wunused-but-set-variable"

#include <cstdlib>  // EXIT_SUCCESS
#include "omp.h"
#include "nvToolsExt.h"
#include "thrust/host_vector.h"
#include "thrust/device_vector.h"

#include <thrust/iterator/counting_iterator.h>
#include "thrust/random.h"

int n_rows;
int n_cols;
int n_nnz;

int* h_indptr;
int* h_indices;
float* h_data;

int* g_indptr;
int* g_indices;
float* g_data;

struct gpu_info {
  hipStream_t stream;
  hipEvent_t  event;
};

std::vector<gpu_info> infos;

hipStream_t master_stream;


struct my_timer_t {
  float time;

  my_timer_t() {
    hipEventCreate(&start_);
    hipEventCreate(&stop_);
    hipEventRecord(start_);
  }

  ~my_timer_t() {
    hipEventDestroy(start_);
    hipEventDestroy(stop_);
  }

  // Alias of each other, start the timer.
  void begin() { hipEventRecord(start_); }
  void start() { this->begin(); }

  float end() {
    hipEventRecord(stop_);
    hipEventSynchronize(stop_);
    hipEventElapsedTime(&time, start_, stop_);

    return milliseconds();
  }

  float seconds() { return time * 1e-3; }
  float milliseconds() { return time; }

 private:
  hipEvent_t start_, stop_;
};

template <typename index_t, typename iterator_t>
void uniform_distribution(index_t begin, index_t end, iterator_t input) {
  using type_t = typename std::iterator_traits<iterator_t>::value_type;

  auto generate_random = [] __device__(int i) -> type_t {
    thrust::default_random_engine rng;
    rng.discard(i);
    return rng();
  };
  
  thrust::transform(thrust::make_counting_iterator(begin), thrust::make_counting_iterator(end), input, generate_random);
}

int get_num_gpus() {
  int num_gpus = -1;
  hipGetDeviceCount(&num_gpus);
  return num_gpus;
}

void enable_peer_access() {
  int num_gpus = get_num_gpus();
  
  for(int i = 0; i < num_gpus; i++) {
    hipSetDevice(i);
    for(int j = 0; j < num_gpus; j++) {
      if(i == j) 
        continue;
      hipDeviceEnablePeerAccess(j, 0);
    }
  }
  
  hipSetDevice(0);
}

void create_contexts() {
  int num_gpus = get_num_gpus();
  
  hipSetDevice(0);
  hipStreamCreateWithFlags(&master_stream, hipStreamNonBlocking);
  
  for(int i = 0 ; i < num_gpus ; i++) {
    gpu_info info;
    hipSetDevice(i);
    hipStreamCreateWithFlags(&info.stream, hipStreamNonBlocking);
    hipEventCreateWithFlags(&info.event, hipEventDisableTiming);
    infos.push_back(info);
  }
  
  hipSetDevice(0);
}

void read_binary(std::string filename) {
  FILE* file = fopen(filename.c_str(), "rb");
  
  auto err = fread(&n_rows, sizeof(int), 1, file);
  err = fread(&n_cols, sizeof(int), 1, file);
  err = fread(&n_nnz,  sizeof(int), 1, file);

  h_indptr  = (int*  )malloc((n_rows + 1) * sizeof(int));
  h_indices = (int*  )malloc(n_nnz        * sizeof(int));
  h_data    = (float*)malloc(n_nnz        * sizeof(float));

  err = fread(h_indptr,  sizeof(int),   n_rows + 1, file);
  err = fread(h_indices, sizeof(int),   n_nnz,      file);
  err = fread(h_data,    sizeof(float), n_nnz,      file);

  hipMallocManaged(&g_indptr,  (n_rows + 1) * sizeof(int));
  hipMallocManaged(&g_indices, n_nnz        * sizeof(int));
  hipMallocManaged(&g_data,    n_nnz        * sizeof(float));

  hipMemcpy(g_indptr, h_indptr, (n_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(g_indices, h_indices, n_nnz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(g_data, h_data, n_nnz * sizeof(int), hipMemcpyHostToDevice);

  // hipMemAdviseSetReadMostly: The device argument is ignored for this advice.
  hipMemAdvise(g_indptr, (n_rows + 1) * sizeof(int), hipMemAdviseSetReadMostly, 0);
  hipMemAdvise(g_indices, n_nnz * sizeof(int), hipMemAdviseSetReadMostly, 0);
  hipMemAdvise(g_data, n_nnz * sizeof(float), hipMemAdviseSetReadMostly, 0);

  int num_gpus = get_num_gpus();

  // Prefetch the graph data to all devices:
  #pragma omp parallel for num_threads(num_gpus)
  for(int i = 0; i < num_gpus; i++) {
    hipMemAdvise(g_indptr, (n_rows + 1) * sizeof(int), hipMemAdviseSetAccessedBy, i);
    hipMemAdvise(g_indices, n_nnz * sizeof(int), hipMemAdviseSetAccessedBy, i);
    hipMemAdvise(g_data, n_nnz * sizeof(float), hipMemAdviseSetAccessedBy, i);
    
    hipMemPrefetchAsync(g_indptr, (n_rows + 1) * sizeof(int), i, 0);
    hipMemPrefetchAsync(g_indices, n_nnz * sizeof(int), i, 0);
    hipMemPrefetchAsync(g_data, n_nnz * sizeof(float), i, 0);
  }
}

void do_test() {
  srand(123123123);
  
  int num_gpus = get_num_gpus();

  // --
  // initialize frontier
  
  thrust::host_vector<int> h_input(n_rows);
  // thrust::host_vector<int> h_output(n_rows);
  for(int i = 0; i < n_rows; i++) h_input[i] = i;
  // for(int i = 0; i < n_rows; i++) h_output[i] = -1;

  // thrust::device_vector<int> input   = h_input;
  // thrust::device_vector<int> output  = h_output;
  int* input;
  hipMallocManaged(&input, n_rows * sizeof(int));
  hipMemcpy(input, h_input.data(), n_rows * sizeof(int), hipMemcpyHostToDevice);
  
  
  // --
  // initialize data structures
  int* colors;
  hipMallocManaged(&colors, n_rows * sizeof(int));
  thrust::fill(thrust::device, colors, colors + n_rows, -1);

  int* h_randoms = (int*)malloc(n_rows * sizeof(int));
  for(int i = 0; i < n_rows; i++) h_randoms[i] = rand() % n_rows;
  
  int* randoms;
  hipMallocManaged(&randoms, n_rows * sizeof(int));
  hipMemcpy(randoms, h_randoms, n_rows * sizeof(int), hipMemcpyHostToDevice);

  // === hipMemAdviseSetReadMostly: The device argument is ignored for this advice.
  hipMemAdvise(randoms, n_rows * sizeof(int), hipMemAdviseSetReadMostly, 0);

  int partitioned = (n_rows + num_gpus - 1) / num_gpus;
  
  // Prefetch the arrays to all devices:
  #pragma omp parallel for num_threads(num_gpus)
  for(int i = 0; i < num_gpus; i++) {
     // === Duplicates the randoms array as read-only on all GPUs.
    hipMemPrefetchAsync(randoms, n_rows * sizeof(int), i, 0);
    hipMemPrefetchAsync(colors, n_rows * sizeof(int), i, 0);
    hipMemPrefetchAsync(input, n_rows * sizeof(int), i, 0);

    // int color_begin = partitioned * i;
    // int color_end = partitioned * (i + 1);

    // === Pin portions of the memory of color/input array to each GPU.
    // a system containing multiple GPUs with peer-to-peer access enabled, 
    // where the data located on one GPU is occasionally accessed by other GPUs. 
    // In such scenarios, migrating data over to the other GPUs is not as 
    // important because the accesses are infrequent and the overhead of migration 
    // may be too high. But preventing faults can still help improve performance, 
    // and so having a mapping set up in advance is useful. 
    // if(i == num_gpus - 1) partitioned = n_rows - (partitioned * (num_gpus - 1));

    hipMemAdvise(colors + partitioned * i, partitioned * sizeof(int), hipMemAdviseSetPreferredLocation, i);
    hipMemAdvise(input + partitioned * i, partitioned * sizeof(int), hipMemAdviseSetPreferredLocation, i);

    // hipMemAdvise(colors + partitioned * i, partitioned * sizeof(int), hipMemAdviseSetAccessedBy, i);
    // hipMemAdvise(input + partitioned * i, partitioned * sizeof(int), hipMemAdviseSetAccessedBy, i);

    // hipMemAdvise(colors, n_rows * sizeof(int), hipMemAdviseSetAccessedBy, i);
    // hipMemAdvise(input, n_rows * sizeof(int), hipMemAdviseSetAccessedBy, i);

    // === Prefetch each portion ahead of time.
    // hipMemPrefetchAsync(colors + partitioned * i, partitioned * sizeof(int), i, 0);
    // hipMemPrefetchAsync(input + partitioned * i, partitioned * sizeof(int), i, 0);

  }

  // --
  // Run
  
  for(int i = 0; i < num_gpus; i++) {
    hipSetDevice(i);  
    hipDeviceSynchronize();
  }

  hipSetDevice(0);

  my_timer_t t;
  std::vector<float> per_iteration_times;
  
  nvtxRangePushA("thrust_work");

  int* indptr = g_indptr;
  int* indices = g_indices;
  float* data = g_data;
  
  int iteration = 0;
  while(iteration < 29) {
    t.begin();

    auto fn = [indptr, indices, data, colors, randoms, iteration] __host__ __device__(int const& vertex) {
      if(vertex == -1) return -1;
      
      int start  = indptr[vertex];
      int end    = indptr[vertex + 1];
      int degree = end - start;

      bool colormax = true;
      bool colormin = true;
      int color     = iteration * 2;

      for (int i = 0; i < degree; i++) {
        int u = indices[start + i];

        if (colors[u] != -1 && (colors[u] != color + 1) && (colors[u] != color + 2) || (vertex == u))
          continue;
        if (randoms[vertex] <= randoms[u])
          colormax = false;
        if (randoms[vertex] >= randoms[u])
          colormin = false;

        if(!colormax && !colormin) return vertex; // optimization
      }

      if (colormax) {
        colors[vertex] = color + 1;
        return -1;
      } else if (colormin) {
        colors[vertex] = color + 2;
        return -1;
      } else {
        return vertex;
      }
    };

    int chunk_size  = (n_rows + num_gpus - 1) / num_gpus;
    
    #pragma omp parallel for num_threads(num_gpus)
    for(int i = 0 ; i < num_gpus ; i++) {
      
      hipSetDevice(i);
      
      auto input_begin  = input + chunk_size * i;
      auto input_end    = input + chunk_size * (i + 1);
      if(i == num_gpus - 1) input_end = input + n_rows;

      thrust::transform(
        thrust::cuda::par.on(infos[i].stream),
        input_begin,
        input_end,
        input_begin,
        fn
      );

      hipEventRecord(infos[i].event, infos[i].stream);
    }
    
    for(int i = 0; i < num_gpus; i++)
      hipStreamWaitEvent(master_stream, infos[i].event, 0);

    hipStreamSynchronize(master_stream);
      
    iteration++;
    t.end();
    per_iteration_times.push_back(t.milliseconds());
    std::cout << t.milliseconds() << std::endl;
  }
  nvtxRangePop();
  
  // Log
  thrust::host_vector<int> out(colors, colors + n_rows);
  thrust::copy(out.begin(), out.begin() + 32, std::ostream_iterator<int>(std::cout, " "));
  std::cout << std::endl;
  
  hipSetDevice(0);

  float total_elapsed = 0;
  for (auto& n : per_iteration_times)
    total_elapsed += n;

  std::cout << "total_elapsed: " << total_elapsed << std::endl;
}

int main(int argc, char** argv) {
  std::string inpath = argv[1];
  
  enable_peer_access();
  create_contexts();
  read_binary(inpath);

  int num_gpus = get_num_gpus();
  std::cout << "color | num_gpus: " << num_gpus << " vertices: " << n_rows << std::endl;

  int num_iters = 4;
  for(int i = 0; i < num_iters; i++)
    do_test();
  
  std::cout << "-----" << std::endl;
  return EXIT_SUCCESS;
}
