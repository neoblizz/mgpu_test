#pragma GCC diagnostic ignored "-Wunused-result"
#pragma GCC diagnostic ignored "-Wunused-but-set-variable"

#include <cstdlib>  // EXIT_SUCCESS
#include "omp.h"
#include "nvToolsExt.h"
#include "thrust/host_vector.h"
#include "thrust/device_vector.h"

#include <thrust/iterator/counting_iterator.h>
#include "thrust/random.h"

#define MANAGED

int n_rows;
int n_cols;
int n_nnz;

int* h_indptr;
int* h_indices;
float* h_data;

int* g_indptr;
int* g_indices;
float* g_data;

struct gpu_info {
  hipStream_t stream;
  hipEvent_t  event;
};

std::vector<gpu_info> infos;

hipStream_t master_stream;


struct my_timer_t {
  float time;

  my_timer_t() {
    hipEventCreate(&start_);
    hipEventCreate(&stop_);
    hipEventRecord(start_);
  }

  ~my_timer_t() {
    hipEventDestroy(start_);
    hipEventDestroy(stop_);
  }

  // Alias of each other, start the timer.
  void begin() { hipEventRecord(start_); }
  void start() { this->begin(); }

  float end() {
    hipEventRecord(stop_);
    hipEventSynchronize(stop_);
    hipEventElapsedTime(&time, start_, stop_);

    return milliseconds();
  }

  float seconds() { return time * 1e-3; }
  float milliseconds() { return time; }

 private:
  hipEvent_t start_, stop_;
};

template <typename index_t, typename iterator_t>
void uniform_distribution(index_t begin, index_t end, iterator_t input) {
  using type_t = typename std::iterator_traits<iterator_t>::value_type;

  auto generate_random = [] __device__(int i) -> type_t {
    thrust::default_random_engine rng;
    rng.discard(i);
    return rng();
  };
  
  thrust::transform(thrust::make_counting_iterator(begin), thrust::make_counting_iterator(end), input, generate_random);
}

int get_num_gpus() {
  int num_gpus = -1;
  hipGetDeviceCount(&num_gpus);
  return num_gpus;
}

void enable_peer_access() {
  int num_gpus = get_num_gpus();
  
  for(int i = 0; i < num_gpus; i++) {
    hipSetDevice(i);
    for(int j = 0; j < num_gpus; j++) {
      if(i == j) 
        continue;
      hipDeviceEnablePeerAccess(j, 0);
    }
  }
  
  hipSetDevice(0);
}

void create_contexts() {
  int num_gpus = get_num_gpus();
  
  hipSetDevice(0);
  hipStreamCreateWithFlags(&master_stream, hipStreamNonBlocking);
  
  for(int i = 0 ; i < num_gpus ; i++) {
    gpu_info info;
    hipSetDevice(i);
    hipStreamCreateWithFlags(&info.stream, hipStreamNonBlocking);
    hipEventCreateWithFlags(&info.event, hipEventDisableTiming);
    infos.push_back(info);
  }
  
  hipSetDevice(0);
}

void read_binary(std::string filename) {
  FILE* file = fopen(filename.c_str(), "rb");
  
  auto err = fread(&n_rows, sizeof(int), 1, file);
  err = fread(&n_cols, sizeof(int), 1, file);
  err = fread(&n_nnz,  sizeof(int), 1, file);

  h_indptr  = (int*  )malloc((n_rows + 1) * sizeof(int));
  h_indices = (int*  )malloc(n_nnz        * sizeof(int));
  h_data    = (float*)malloc(n_nnz        * sizeof(float));

  err = fread(h_indptr,  sizeof(int),   n_rows + 1, file);
  err = fread(h_indices, sizeof(int),   n_nnz,      file);
  err = fread(h_data,    sizeof(float), n_nnz,      file);

#ifdef MANAGED
  hipMallocManaged(&g_indptr,  (n_rows + 1) * sizeof(int));
  hipMallocManaged(&g_indices, n_nnz        * sizeof(int));
  hipMallocManaged(&g_data,    n_nnz        * sizeof(float));
#else
  hipMalloc(&g_indptr, (n_rows + 1) * sizeof(int));
  hipMalloc(&g_indices, n_nnz       * sizeof(int));
  hipMalloc(&g_data,    n_nnz       * sizeof(float));
#endif

  hipMemcpy(g_indptr, h_indptr, (n_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(g_indices, h_indices, (n_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(g_data, h_data, (n_rows + 1) * sizeof(int), hipMemcpyHostToDevice);

#ifdef MANAGED

  // hipMemAdviseSetReadMostly: The device argument is ignored for this advice.
  hipMemAdvise(g_indptr, (n_rows + 1) * sizeof(int), hipMemAdviseSetReadMostly, 0);
  hipMemAdvise(g_indices, n_nnz * sizeof(int), hipMemAdviseSetReadMostly, 0);
  hipMemAdvise(g_data, n_nnz * sizeof(float), hipMemAdviseSetReadMostly, 0);

  int num_gpus = get_num_gpus();

  // Prefetch the graph data to all devices:
  #pragma omp parallel for num_threads(num_gpus)
  for(int i = 0; i < num_gpus; i++) {
    hipMemPrefetchAsync(g_indptr, (n_rows + 1) * sizeof(int), i, 0);
    hipMemPrefetchAsync(g_indices, n_nnz * sizeof(int), i, 0);
    hipMemPrefetchAsync(g_data, n_nnz * sizeof(float), i, 0);
  }

#endif  
}

void do_test() {
  srand(123123123);
  
  int num_gpus = get_num_gpus();

  // --
  // initialize frontier
  
  thrust::host_vector<int> h_input(n_rows);
  thrust::host_vector<int> h_output(n_rows);
  for(int i = 0; i < n_rows; i++) h_input[i] = i;
  for(int i = 0; i < n_rows; i++) h_output[i] = -1;

  thrust::device_vector<int> input   = h_input;
  thrust::device_vector<int> output  = h_output;
  
  // --
  // initialize data structures
  int* colors;
  hipMallocManaged(&colors, n_rows * sizeof(int));
  thrust::fill(thrust::device, colors, colors + n_rows, -1);

  int* h_randoms = (int*)malloc(n_rows * sizeof(int));
  for(int i = 0; i < n_rows; i++) h_randoms[i] = rand() % n_rows;
  
  int* randoms;
  hipMallocManaged(&randoms, n_rows * sizeof(int));
  hipMemcpy(randoms, h_randoms, n_rows * sizeof(int), hipMemcpyHostToDevice);

#ifdef MANAGED
  // === hipMemAdviseSetReadMostly: The device argument is ignored for this advice.
  hipMemAdvise(randoms, n_rows * sizeof(int), hipMemAdviseSetReadMostly, 0);

  int partitioned_color = n_rows + num_gpus - 1 / num_gpus;
  
  // Prefetch the arrays to all devices:
  #pragma omp parallel for num_threads(num_gpus)
  for(int i = 0; i < num_gpus; i++) {
     // === Duplicates the randoms array as read-only on all GPUs.
    hipMemPrefetchAsync(randoms, n_rows * sizeof(int), i, 0);

    // int color_begin = partitioned_color * i;
    // int color_end = partitioned_color * (i + 1);

    // === Pin portions of the memory of color array to each GPU.
    // a system containing multiple GPUs with peer-to-peer access enabled, 
    // where the data located on one GPU is occasionally accessed by other GPUs. 
    // In such scenarios, migrating data over to the other GPUs is not as 
    // important because the accesses are infrequent and the overhead of migration 
    // may be too high. But preventing faults can still help improve performance, 
    // and so having a mapping set up in advance is useful. 
    if(i == num_gpus - 1) partitioned_color = n_rows - (partitioned_color * (num_gpus - 1));
    hipMemAdvise(colors + partitioned_color * i, partitioned_color * sizeof(int), hipMemAdviseSetAccessedBy, i);
    hipMemAdvise(colors + partitioned_color * i, partitioned_color * sizeof(int), hipMemAdviseSetPreferredLocation, i);

    // === Prefetch each portion ahead of time.
    hipMemPrefetchAsync(colors + partitioned_color * i, partitioned_color * sizeof(int), i, 0);
  }

#endif

  // --
  // Run
  
  for(int i = 0; i < num_gpus; i++) {
    hipSetDevice(i);  
    hipDeviceSynchronize();
  }

  hipSetDevice(0);

  my_timer_t t;
  t.begin();

  int new_sizes[num_gpus];
  
  int* indptr  = g_indptr;
  int* indices = g_indices;
  float* data  = g_data;
  
  nvtxRangePushA("thrust_work");
  
  int iteration = 0;
  while(input.size() > 4) {

    auto fn = [indptr, indices, data, colors, randoms, iteration] __host__ __device__(int const& vertex) -> bool {
        int start  = indptr[vertex];
        int end    = indptr[vertex + 1];
        int degree = end - start;

        bool colormax = true;
        bool colormin = true;
        int color     = iteration * 2;

        for (int i = 0; i < degree; i++) {
          int u = indices[start + i];

          if (colors[u] != -1 && (colors[u] != color + 1) && (colors[u] != color + 2) || (vertex == u))
            continue;
          if (randoms[vertex] <= randoms[u])
            colormax = false;
          if (randoms[vertex] >= randoms[u])
            colormin = false;
        }

        if (colormax) {
          colors[vertex] = color + 1;
          return false;
        } else if (colormin) {
          colors[vertex] = color + 2;
          return false;
        } else {
          return true;
        }
      };

    int chunk_size  = (input.size() + num_gpus - 1) / num_gpus;
    
    #pragma omp parallel for num_threads(num_gpus)
    for(int i = 0 ; i < num_gpus ; i++) {
      
      hipSetDevice(i);
      
      auto input_begin  = input.begin() + chunk_size * i;
      auto input_end    = input.begin() + chunk_size * (i + 1);
      auto output_begin = output.begin() + chunk_size * i;
      if(i == num_gpus - 1) input_end = input.end();
      
      auto new_output_end = thrust::copy_if(
        thrust::cuda::par.on(infos[i].stream),
        input_begin,
        input_end,
        output_begin,
        fn
      );
      new_sizes[i] = (int)thrust::distance(output_begin, new_output_end);
      hipEventRecord(infos[i].event, infos[i].stream);
    }
    
    for(int i = 0; i < num_gpus; i++)
      hipStreamWaitEvent(master_stream, infos[i].event, 0);

    hipStreamSynchronize(master_stream);
    
    int total_length = 0;
    int offsets[num_gpus];
    offsets[0] = 0;
    for(int i = 1 ; i < num_gpus ; i++) offsets[i] = new_sizes[i - 1] + offsets[i - 1];
    for(int i = 0 ; i < num_gpus ; i++) total_length += new_sizes[i];

    // Reduce
    hipSetDevice(0);
    for(int i = 0; i < num_gpus; i++) {
      auto output_begin = output.begin() + chunk_size * i;
      thrust::copy_n(
        thrust::cuda::par.on(infos[0].stream),
        output_begin, 
        new_sizes[i], 
        input.begin() + offsets[i]
      );
    }
    
    hipEventRecord(infos[0].event, infos[0].stream);
    hipStreamWaitEvent(master_stream, infos[0].event, 0);
    hipStreamSynchronize(master_stream);
    
    input.resize(total_length);
    output.resize(total_length);
      
    iteration++;
    t.end();  
    std::cout << "elapsed: " << t.milliseconds() << std::endl;
  }
  nvtxRangePop();
  
  // Log
  thrust::host_vector<int> out(colors, colors + n_rows);
  thrust::copy(out.begin(), out.begin() + 32, std::ostream_iterator<int>(std::cout, " "));
  std::cout << std::endl;
  
  hipSetDevice(0);
  t.end();  
  std::cout << "total_elapsed: " << t.milliseconds() << std::endl;
}

int main(int argc, char** argv) {
  std::string inpath = argv[1];
  
  enable_peer_access();
  create_contexts();
  read_binary(inpath);

  int num_gpus = get_num_gpus();
  std::cout << "color | num_gpus: " << num_gpus << " vertices: " << n_rows << std::endl;

  int num_iters = 4;
  for(int i = 0; i < num_iters; i++)
    do_test();
  
  std::cout << "-----" << std::endl;
  return EXIT_SUCCESS;
}